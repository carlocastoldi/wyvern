#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <stdint.h>
#include <sys/time.h>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 128
#endif
#ifndef BLOCK_COUNT
#define BLOCK_COUNT 128
#endif

__global__ void matmul(const unsigned n, const unsigned* a, const unsigned* b, unsigned* c) {
    const unsigned tsize = blockDim.x * gridDim.x;
    for(unsigned tid = blockDim.x * blockIdx.x + threadIdx.x; tid < n * n; tid += tsize) {
        const unsigned i = tid % n;
        const unsigned j = tid / n;
        unsigned acc = 0;
        for(unsigned k = 0; k < n; k++) {
            acc += a[i * n + k] * b[k * n + j];
        }
        c[i * n + j] = acc;
    }
}

int main(int argc, char* argv[]) {
    assert(argc == 2);
    freopen(argv[1], "r", stdin);
    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    unsigned* h_a,* h_b,* h_c,* d_a,* d_b,* d_c;
    unsigned n, m, trace = 0;
    scanf("%u%u", &n, &m);
    h_a = (unsigned*)calloc(n * n, sizeof(unsigned));
    h_b = (unsigned*)calloc(n * n, sizeof(unsigned));
    h_c = (unsigned*)calloc(n * n, sizeof(unsigned));
    for(unsigned i = 0; i < m; i++) {
        unsigned a, b;
        scanf("%u%u", &a, &b);
        h_a[a * n + b] = h_b[a * n + b] = 1;
        h_a[b * n + a] = h_b[b * n + a] = 1;
    }
    hipMalloc(&d_a, n * n * sizeof(unsigned));
    hipMalloc(&d_b, n * n * sizeof(unsigned));
    hipMalloc(&d_c, n * n * sizeof(unsigned));
    hipMemcpy(d_a, h_a, n * n * sizeof(unsigned), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, n * n * sizeof(unsigned), hipMemcpyHostToDevice);
    hipMemcpy(d_c, h_c, n * n * sizeof(unsigned), hipMemcpyHostToDevice);
    matmul<<<BLOCK_COUNT, BLOCK_SIZE>>>(n, d_a, d_b, d_c);
    matmul<<<BLOCK_COUNT, BLOCK_SIZE>>>(n, d_a, d_c, d_b);
    hipMemcpy(h_c, d_b, n * n * sizeof(unsigned), hipMemcpyDeviceToHost);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    for(unsigned i = 0; i < n; i++) {
        trace += h_c[i * n + i];
    }
    free(h_a);
    free(h_b);
    free(h_c);
    assert(trace % 6 == 0);
    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    uint64_t delta_us = (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_nsec - start.tv_nsec) / 1000;
    double delta = double(delta_us) / 1e6;
    printf("%.9lf\n", delta);
    fprintf(stderr, "Triangles: %u\n", trace / 6);
    return 0;
}
