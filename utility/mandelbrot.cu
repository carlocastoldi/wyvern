#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <stdint.h>
#include <sys/time.h>

#define ITERATIONS 2000
#define WIDTH 3840
#define HEIGHT 2160
#define CENTER_X -0.75
#define CENTER_Y 0.0
#define ZOOM (float(HEIGHT) / 2.5)

__global__ void mandelbrot(unsigned* dim, float* output) {
    unsigned width = dim[0];
    unsigned height = dim[1];
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    for(; tid < width * height; tid += blockDim.x * gridDim.x) {
        float x = tid % width;
        float y = tid / width;
        x -= width / 2.0;
        y -= height / 2.0;
        x /= ZOOM;
        y /= ZOOM;
        x += CENTER_X;
        y += CENTER_Y;
        float a = 0.0, b = 0.0;
        for(unsigned i = 0; i < ITERATIONS; i++) {
            float tmp_a = a * a - b * b + x;
            b = 2.0 * a * b + y;
            a = tmp_a;
        }
        output[tid] = a * a + b * b;
    }
}

int main() {
    unsigned* host_dim;
    float* host_output;
    unsigned* device_dim;
    float* device_output;
    struct timespec start, end;
    host_dim = (unsigned*)malloc(2 * sizeof(unsigned));
    assert(host_dim);
    host_output = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
    assert(host_output);
    hipMalloc(&device_dim, 2 * sizeof(unsigned));
    hipMalloc(&device_output, WIDTH * HEIGHT * sizeof(float));
    host_dim[0] = WIDTH;
    host_dim[1] = HEIGHT;
    hipMemcpy(device_dim, host_dim, 2 * sizeof(unsigned), hipMemcpyHostToDevice);
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    mandelbrot<<<(WIDTH * HEIGHT + 256) / 256, 256>>>(device_dim, device_output);
    hipDeviceSynchronize();
    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    hipMemcpy(host_output, device_output, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(device_output);
    hipFree(device_dim);
    FILE* output = fopen("out.ppm", "w");
    fprintf(output, "P2\n%u %u\n255\n", host_dim[0], host_dim[1]);
    for(unsigned i = 0; i < WIDTH * HEIGHT; i++) {
        fprintf(output, "%d\n", (host_output[i] <= 2.0) ? (0) : (255));
    }
    fclose(output);
    free(host_dim);
    free(host_output);
    uint64_t delta_us = (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_nsec - start.tv_nsec) / 1000;
    double delta = double(delta_us) / 1e6;
    printf("Time: %.3lf\n", delta);
    return 0;
}
