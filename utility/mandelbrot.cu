#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <stdint.h>
#include <sys/time.h>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 128
#endif
#ifndef BLOCK_COUNT
#define BLOCK_COUNT 128
#endif
#define CENTER_X -0.75
#define CENTER_Y 0.0
#define ZOOM (float(height) / 2.5)

__global__ void mandelbrot(unsigned* dim, float* output, unsigned iterations) {
    unsigned width = dim[0];
    unsigned height = dim[1];
    unsigned tid = blockDim.x * blockIdx.x + threadIdx.x;
    for(; tid < width * height; tid += blockDim.x * gridDim.x) {
        float x = tid % width;
        float y = tid / width;
        x -= width / 2.0;
        y -= height / 2.0;
        x /= ZOOM;
        y /= ZOOM;
        x += CENTER_X;
        y += CENTER_Y;
        float a = 0.0, b = 0.0;
        for(unsigned i = 0; i < iterations; i++) {
            float tmp_a = a * a - b * b + x;
            b = 2.0 * a * b + y;
            a = tmp_a;
        }
        output[tid] = a * a + b * b;
    }
}

int main(int argc, char* argv[]) {
    assert(argc == 4);
    unsigned WIDTH, HEIGHT, ITERATIONS;
    WIDTH = atoi(argv[1]);
    HEIGHT = atoi(argv[2]);
    ITERATIONS = atoi(argv[3]);
    unsigned* host_dim;
    float* host_output;
    unsigned* device_dim;
    float* device_output;
    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC_RAW, &start);
    host_dim = (unsigned*)malloc(2 * sizeof(unsigned));
    assert(host_dim);
    host_output = (float*)malloc(WIDTH * HEIGHT * sizeof(float));
    assert(host_output);
    hipMalloc(&device_dim, 2 * sizeof(unsigned));
    hipMalloc(&device_output, WIDTH * HEIGHT * sizeof(float));
    host_dim[0] = WIDTH;
    host_dim[1] = HEIGHT;
    hipMemcpy(device_dim, host_dim, 2 * sizeof(unsigned), hipMemcpyHostToDevice);
    mandelbrot<<<BLOCK_COUNT, BLOCK_SIZE>>>(device_dim, device_output, ITERATIONS);
    hipMemcpy(host_output, device_output, WIDTH * HEIGHT * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(device_output);
    hipFree(device_dim);
    clock_gettime(CLOCK_MONOTONIC_RAW, &end);
    FILE* output = fopen("out.ppm", "w");
    fprintf(output, "P2\n%u %u\n255\n", host_dim[0], host_dim[1]);
    for(unsigned i = 0; i < WIDTH * HEIGHT; i++) {
        fprintf(output, "%d\n", (host_output[i] <= 2.0) ? (0) : (255));
    }
    fclose(output);
    free(host_dim);
    free(host_output);
    uint64_t delta_us = (end.tv_sec - start.tv_sec) * 1000000 + (end.tv_nsec - start.tv_nsec) / 1000;
    double delta = double(delta_us) / 1e6;
    printf("%.9lf\n", delta);
    return 0;
}
